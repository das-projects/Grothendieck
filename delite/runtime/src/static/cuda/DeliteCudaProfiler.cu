#include "DeliteCudaProfiler.h"

static BufferedFileWriter** profileWriters = new BufferedFileWriter*[DELITE_NUM_CUDA];
static std::stack< cudatimer_t >** timermaps = new std::stack< cudatimer_t >*[DELITE_NUM_CUDA];
static double appStartTime = 0;

static std::string profileFilePrefix;

double milliseconds(struct timeval t) {
  return double(t.tv_sec * 1000) + (double(t.tv_usec) / 1000);
}

void InitDeliteCudaTimer(int32_t tid, int32_t lowestCudaTid) {
  if (tid == 0) {
	struct timeval a;
    gettimeofday(&a,NULL);
    appStartTime = milliseconds(a);

    char* cwd = get_current_dir_name();
    std::stringstream ss;
    ss << cwd << "/profile/profile_t_";
    profileFilePrefix = ss.str();
  }

  timermaps[tid] = new std::stack< cudatimer_t >();
  std::stringstream ss; 
  ss << profileFilePrefix << ( lowestCudaTid + tid ) << ".csv";
  profileWriters[tid] = new BufferedFileWriter(ss.str().c_str());
}

void DeliteCudaTimerStart(int32_t tid, std::string name) {
  hipDeviceSynchronize();

  struct timeval start;
  gettimeofday(&start,NULL);
  cudatimer_t timer = {start};

  timermaps[tid]->push(timer);
}

void DeliteCudaTimerStop(int32_t tid, std::string name, bool isMultiLoop) {
  hipDeviceSynchronize();

  struct timeval stop;
  gettimeofday(&stop,NULL);

  double start = milliseconds( timermaps[tid]->top().start ); 
  double end = milliseconds( stop );
  double elapsedMillis = end - start;

  timermaps[tid]->pop();
  profileWriters[tid]->writeTimer( name, long(start - appStartTime), elapsedMillis, timermaps[tid]->size(), tid, isMultiLoop );
}

#ifndef __DELITE_CPP_PROFILER_H__ 

BufferedFileWriter::BufferedFileWriter(const char* fileName)
{
    fs.open(fileName);
}

void BufferedFileWriter::writeTimer(std::string kernel, long start, double duration, int32_t level, int32_t tid, bool isMultiLoop) {
  if (isMultiLoop) {
    fs << kernel << "_" << tid << "," << start << "," << duration << "," << level << std::endl;
  } else {
    fs << kernel << "," << start << "," << duration << "," << level << std::endl;
  }
}

void BufferedFileWriter::close() {
  fs.close();
}

#endif

void DeliteCudaTimerClose(int32_t tid, int32_t rid, JNIEnv* env) {
  for (int32_t i = 0; i < DELITE_NUM_CUDA; i++) {
    profileWriters[i]->close();
  }
}
